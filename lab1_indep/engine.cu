#include <iostream>
#include <hip/hip_runtime.h>

#define EPS     1e-4f
#define PI      3.14159265359f

#define gpuErrChk(func) { gpuAssert((func), __FILE__, __LINE__); }
inline void gpuAssert(
    hipError_t ret,
    const char *fname,
    int line,
    bool forceStop=true
) {
    if (ret != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(ret);
        std::cerr << ' ' << fname;
        std::cerr << " ln" << line << std::endl;
        if (forceStop) {
            exit(ret);
        }
    }
}

int main() {
    const int width = 320, height = 240;

    float3 *h_frame = new float3[width*height];
    float3 *d_frame;
    gpuErrChk(hipMalloc(&d_frame, width*height * sizeof(float3)));

    gpuErrChk(hipFree(d_frame));
    delete[] h_frame;

    return 0;
}
