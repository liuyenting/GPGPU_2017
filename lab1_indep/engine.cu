#include "hip/hip_runtime.h"
// Library
#include <hip/hip_runtime.h>

// Internal
#include "cutil_math.h"

// System
#include <iostream>

#define EPS     1e-4f
#define PI      3.14159265359f

#define gpuErrChk(func) { gpuAssert((func), __FILE__, __LINE__); }
inline void gpuAssert(
    hipError_t ret,
    const char *fname,
    int line,
    bool forceStop=true
) {
    if (ret != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(ret);
        std::cerr << ' ' << fname;
        std::cerr << " ln" << line << std::endl;
        if (forceStop) {
            exit(ret);
        }
    }
}


struct Ray {
    float3 orig;    // origin
    float3 dir;     // direction

    __device__
    Ray(const float3 _orig, const float3 _dir)
        : orig(_orig), dir(_dir) {
    }
};

/*
 * Reflection type
 */
enum Refl_t {
    DIFF = 0,   // diffuse
    SPEC,       // speckle
    REFR        // refract
};

struct Object {
    float3 pos;     // position in world coordinate
    float3 emi;     // RGB emission
    float3 color;   // RGB
    Refl_t refl;    // type of reflection

    __device__
    virtual bool distance(const Ray &ray) const = 0;
    __device__
    virtual bool distance(const Object &obj) const = 0;
};

struct Sphere : Object {
    float rad;  // radius

    __device__
    virtual bool distance(const Ray &ray) const {
        /*
         * ray equation: p(x, y, z) = ray.orig + t*ray.dir
         * sphere equation: x^2 + y^2 + z^2 = rad^2
         *
         * quadratic: ax^2 + bx + c = 0 -> x = (-b +- sqrt(b^2 - 3ac)) / 2a
         *
         * solve t^2*ray.dir*ray.dir + 2*t*(orig-p)*ray.dir + (orig-p)*(orig-p) - rad*rad = 0
         */
        float3 dist = pos - ray.orig;
        float b = dot(dist, ray.dir);

        // discriminant
        float disc = b*b - dot(dist, dist) + rad*rad;

        if (disc < 0) {
            // ignroe complex solution
            return 0;
        } else {
            disc = sqrtf(disc);
        }
        // return the closest point relative the the origin of light ray
        float t;
        return ((t = b - disc) > EPS) ? t : ( ((t = b + disc) > EPS) ? t : 0 );
    }
};

int main() {
    const int width = 320, height = 240;

    float3 *h_frame = new float3[width*height];
    float3 *d_frame;
    gpuErrChk(hipMalloc(&d_frame, width*height * sizeof(float3)));

    gpuErrChk(hipFree(d_frame));
    delete[] h_frame;

    return 0;
}
