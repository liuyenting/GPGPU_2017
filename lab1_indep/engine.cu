#include "hip/hip_runtime.h"
// Library
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>

// Internal
#include "cutil_math.h"

// System
#include <iostream>
#include <sstream>      // stringstream
#include <fstream>      // ofstream

#define EPS     1e-4f

#define gpuErrChk(func) { gpuAssert((func), __FILE__, __LINE__); }
inline void gpuAssert(
    hipError_t ret,
    const char *fname,
    int line,
    bool forceStop=true
) {
    if (ret != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(ret);
        std::cerr << ' ' << fname;
        std::cerr << " ln" << line << std::endl;
        if (forceStop) {
            exit(ret);
        }
    }
}

#define DIVUP(a, b) ((a+b-1)/b)

struct Ray {
    float3 orig;    // origin
    float3 dir;     // direction

    __device__
    Ray() {
    }

    __host__ __device__
    Ray(const float3 _orig, const float3 _dir)
        : orig(_orig), dir(_dir) {
    }
};

/*
 * Reflection type
 */
enum Refl_t {
    DIFF = 1,   // diffuse
    SPEC,       // speckle
    REFR        // refract
};

struct Sphere {
     float rad;            // radius
     float3 pos, emi, col; // position, emission, colour
     Refl_t refl;          // reflection type (e.g. diffuse)

    __device__
    float distance(const Ray &ray) const {
        /*
         * ray equation: p(x, y, z) = ray.orig + t*ray.dir
         * sphere equation: x^2 + y^2 + z^2 = rad^2
         *
         * quadratic: ax^2 + bx + c = 0 -> x = (-b +- sqrt(b^2 - 3ac)) / 2a
         *
         * solve t^2*ray.dir*ray.dir + 2*t*(orig-p)*ray.dir + (orig-p)*(orig-p) - rad*rad = 0
         */
         float3 dist = pos - ray.orig;
         float b = dot(dist, ray.dir);

         // discriminant
         float disc = b*b - dot(dist, dist) + rad*rad;

        if (disc < 0) {
            // ignroe complex solution
            return 0;
        } else {
            disc = sqrtf(disc);
        }
        // return the closest point relative the the origin of light ray
        float t;
        return ((t = b - disc) > EPS) ? t : ( ((t = b + disc) > EPS) ? t : 0 );
     }
};

__constant__
Ray c_camera;
__constant__
Sphere  c_spheres[8];

__device__
inline bool intersect_scene(const Ray &r, float &t, int &id) {
    float n = sizeof(c_spheres) / sizeof(Sphere), d;
    t = HIP_INF_F;  // t is distance to closest intersection, initialise t to a huge number outside scene
    for (int i = int(n); i--;) { // test all scene objects for intersection
        if ((d = c_spheres[i].distance(r)) && (d < t)) {  // if newly computed intersection distance d is smaller than current closest intersection distance
            t = d;  // keep track of distance along ray to closest intersection point
            id = i; // and closest intersected object
        }
    }
    return (t < HIP_INF_F); // returns true if an intersection with the scene occurred, false when no hit
}

// random number generator from https://github.com/gz/rust-raytracer

__device__
static float getrandom(unsigned int *seed0, unsigned int *seed1) {
    *seed0 = 36969 * ((*seed0) & 65535) + ((*seed0) >> 16);  // hash the seeds using bitwise AND and bitshifts
    *seed1 = 18000 * ((*seed1) & 65535) + ((*seed1) >> 16);

    unsigned int ires = ((*seed0) << 16) + (*seed1);

    // Convert to float
    union {
        float f;
        unsigned int ui;
    } res;

    res.ui = (ires & 0x007fffff) | 0x40000000;  // bitwise AND, bitwise OR

    return (res.f - 2.f) / 2.f;
}

/*
 * rendering equation:
 * outgoing radiance at a point = emitted radiance + reflected radiance
 *
 * reflected radiance = sum of (incoming radiance from hemisphere above point)
 *                      * reflectance function of material
 *                      * cosine incident angle
 */
__device__
float3 radiance(Ray ray, unsigned int *s1, unsigned int *s2){
    float3 color = make_float3(0.0f); // accumulated color
    float3 mask = make_float3(1.0f);

    // bounce the ray
    for (int count = 0; count < 4; count++) {
        float t;           // distance to closest intersection
        int id = 0;        // index of closest intersected sphere

        // test whether the scence is intersected
        if (!intersect_scene(ray, t, id)) {
            // return black if missed
            return make_float3(0.0f);
        }

        // compute impact location and normal vector
        const Sphere &obj =  c_spheres[id];
        float3 p = ray.orig + ray.dir * t;  // impact location
        float3 n = normalize(p - obj.pos);  // normal
        // convert to front facing
        n = (dot(n, ray.dir) < 0) ? n : n * (-1);

        // add the photons from current object to accumulate the color
        color += mask * obj.emi;

        /*
         * generate new diffuse ray
         *     .orig = impac location
         *     .dir = random direction above the impact location
         */
        float r1 = 2 * HIP_PI_F * getrandom(s1, s2); // pick random number on unit circle (radius = 1, circumference = 2*Pi) for azimuth
        float r2 = getrandom(s1, s2);  // pick random number for elevation
        float r2s = sqrtf(r2);

        // compute local orthonormal basis uvw at hitpoint to use for calculation random ray direction
        // first vector = normal at hitpoint, second vector is orthogonal to first, third vector is orthogonal to first two vectors
        float3 w = n;
        float3 u = normalize(cross((fabs(w.x) > .1 ? make_float3(0, 1, 0) : make_float3(1, 0, 0)), w));
        float3 v = cross(w, u);

        // compute random ray direction on hemisphere using polar coordinates
        // cosine weighted importance sampling (favours ray directions closer to normal direction)
        float3 d = normalize(u*cos(r1)*r2s + v*sin(r1)*r2s + w*sqrtf(1 - r2));

        // new ray origin is intersection point of previous ray with scene
        ray.orig = p + n*0.05f; // offset ray origin slightly to prevent self intersection
        ray.dir = d;

        mask *= obj.col;    // multiply with colour of object
        mask *= dot(d, n);  // weigh light contribution using cosine of angle between incident light and normal
        mask *= 2;          // fudge factor
    }

    return color;
}

__global__
void renderKernel(
    float3 *frame,
    const int width,
    const int height,
    const int ntrials
) {
    const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if ((x >= width) || (y >= height)) {
        return;
    }

    // calculate the index, y direction flipped
    //unsigned int i = y*width + x;
    unsigned int i = (height - y - 1) * width + x;

    unsigned int s1 = x;  // seeds for random number generator
    unsigned int s2 = y;

    // generate ray directed at lower left corner of the screen
    // compute directions for all other rays by adding cx and cy increments in x and y direction
    float3 cx = make_float3(width * .5135 / height, 0.0f, 0.0f); // ray direction offset in x direction
    float3 cy = normalize(cross(cx, c_camera.dir)) * .5135; // ray direction offset in y direction (.5135 is field of view angle)
    float3 r; // r is final pixel color

    r = make_float3(0.0f);

    // sampling for each pixel
    for (int s = 0; s < ntrials; s++) {
        // compute primary ray direction
        float3 d = c_camera.dir + cx*((.25 + x) / width - .5) + cy*((.25 + y) / height - .5);

        // create primary ray, add incoming radiance to pixelcolor
        r = r + radiance(Ray(c_camera.orig + d * 40, normalize(d)), &s1, &s2)*(1. / ntrials);
                   // Camera rays are pushed ^^^^^^ forward to start in interior
    }

    // write rgb value of pixel to image buffer on the GPU, clamp value to [0.0f, 1.0f] range
    frame[i] = clamp(r, 0.0f, 1.0f);
}

/*
 * 1) convert RGB float from [0, 1] to [0, 255]
 * 2) perform gamma correction
 */
inline int toInt(float x) {
    return int(pow(clamp(x, 0.0f, 1.0f), 1/2.2f) * 255 + .5);
}

__global__
void convertKernel(
    float3 *out,
    const float3 *in,
    const int width,
    const int height
) {
}

/*
 * using 9 spheres to form a cornell box
 *
 * {
 *     float radius,
 *     { float3 position },
 *     { float3 emission },
 *     { float3 color },
 *     Refl_t refl
 * }
 */
Sphere h_spheres[] = {
    { 1e5f, { 1e5f + 1.0f, 40.8f, 81.6f }, { 0.0f, 0.0f, 0.0f }, { 0.75f, 0.25f, 0.25f }, DIFF }, //Left
    { 1e5f, { -1e5f + 99.0f, 40.8f, 81.6f }, { 0.0f, 0.0f, 0.0f }, { .25f, .25f, .75f }, DIFF }, //Right
    { 1e5f, { 50.0f, 40.8f, 1e5f }, { 0.0f, 0.0f, 0.0f }, { .75f, .75f, .75f }, DIFF }, //Back
    { 1e5f, { 50.0f, 40.8f, -1e5f + 600.0f }, { 0.0f, 0.0f, 0.0f }, { 1.00f, 1.00f, 1.00f }, DIFF }, //Frnt
    { 1e5f, { 50.0f, 1e5f, 81.6f }, { 0.0f, 0.0f, 0.0f }, { .75f, .75f, .75f }, DIFF }, //Botm
    { 1e5f, { 50.0f, -1e5f + 81.6f, 81.6f }, { 0.0f, 0.0f, 0.0f }, { .75f, .75f, .75f }, DIFF }, //Top
    { 8.0f, { 50.0f, 40.0f, 78.0f }, { 0.0f, 0.0f, 0.0f }, { 1.0f, 1.0f, 1.0f }, DIFF }, // small sphere 2
    { 600.0f, { 50.0f, 681.6f - .25f, 81.6f }, { 2.0f, 1.8f, 1.6f }, { 0.0f, 0.0f, 0.0f }, DIFF }  // Light
};

void updatePhysics(const float t_step, const int priId) {
    float theta = 2*HIP_PI_F * t_step/5;
    h_spheres[priId].pos = make_float3(
        16.0f * cosf(theta) + 50.0f,
        16.0f * sinf(theta) + 40.0f,
        78.0f
    );
}

int main(){
    const int width = 640, height = 480;
    const int fps = 24, nframes = 1;
    const int ntrials = 2048;

    const float t_step = 1.0f; //1.0f/fps;

    Ray camera(
        make_float3(50, 52, 295.6),
        normalize(make_float3(0, -0.042612, -1))
    );

    const int nelem = width * height;
    const size_t nbytes = nelem * sizeof(float3);

    float3* h_frame = new float3[nelem];
    float3* d_frame;
    gpuErrChk(hipMalloc(&d_frame, nbytes));

    // copy camera position to constant memory
    gpuErrChk(hipMemcpyToSymbol(HIP_SYMBOL(c_camera), &camera, sizeof(Ray)));

    std::cout << "CUDA initialized" << std::endl << std::flush;

    dim3 threads(16, 16);
    dim3 blocks(DIVUP(width, threads.x), DIVUP(height, threads.y));
    std::stringstream ss;
    std::ofstream outfile;
    for (int iframe = 1; iframe <= nframes; iframe++) {
        std::cout << "Frame " << iframe;
        std::cout << ", t=" << (iframe * t_step) << 's' << std::endl;

        std::cout << "\r\tUPDATING...     " << std::flush;

        // update the position
        updatePhysics(t_step * iframe, 6);
        gpuErrChk(hipMemcpyToSymbol(HIP_SYMBOL(c_spheres), &h_spheres, sizeof(h_spheres)));

        std::cout << "\r\tRUNNING...     " << std::flush;

        renderKernel<<<blocks, threads>>>(d_frame, width, height, ntrials);

        // copy the result back
        gpuErrChk(hipMemcpy(h_frame, d_frame, nbytes, hipMemcpyDeviceToHost));

        std::cout << "\r\tSAVING...     " << std::flush;

        ss.str(std::string());
        ss.clear();
        // build new filename
        ss << "frame_" << iframe << ".ppm";

        outfile.open(ss.str());

        // write PPM definition
        outfile << "P3" << std::endl;
        outfile << width << ' ' << height << " 255" << std::endl;
        // write image
        for (int i = 0; i < nelem; i++) {
            outfile << toInt(h_frame[i].x) << ' ';
            outfile << toInt(h_frame[i].y) << ' ';
            outfile << toInt(h_frame[i].z) << ' ';
        }

        outfile.close();

        std::cout << "\r\tDone!     " << std::endl;
    }

    gpuErrChk(hipFree(d_frame));
    delete[] h_frame;

    return 0;
}
