#include "hip/hip_runtime.h"
#include "lab3.h"
#include <cstdio>

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

__global__ void SimpleClone(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
)
{
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt+xt;
	if (yt < ht and xt < wt and mask[curt] > 127.0f) {
		const int yb = oy+yt, xb = ox+xt;
		const int curb = wb*yb+xb;
		if (0 <= yb and yb < hb and 0 <= xb and xb < wb) {
			output[curb*3+0] = target[curt*3+0];
			output[curb*3+1] = target[curt*3+1];
			output[curb*3+2] = target[curt*3+2];
		}
	}
}

__global__ void CalculateFixed(
	const float *background,
	const float *target,
	const float *mask,
	float *fixed,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
) {
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int clt = wt*yt+xt;

	// ignore out-of-range pixels
	if (yt >= ht or xt >= wt) {
		return;
	}

	const int yb = oy+yt, xb = ox+xt;
	const int clb = wb*yb+xb;

	// 1px spacing, using background values
	if (yt < 1 or yt >= ht-1 or xt < 1 or xt >= wt-1) {
		fixed[clt*3+0] = background[clb*3+0];
		fixed[clt*3+1] = background[clb*3+1];
		fixed[clt*3+2] = background[clb*3+2];
		return;
	}

	// calculate target N, S, W, E linear index
	const int nlt = wt*(yt+1)+xt;
	const int slt = wt*(yt-1)+xt;
	const int wlt = wt*yt+(xt-1);
	const int elt = wt*yt+(xt+1);
	// calculate background N, S, W, E linear index
	const int nlb = wb*(yb+1)+xb;
	const int slb = wb*(yb-1)+xb;
	const int wlb = wb*yb+(xb+1);
	const int elb = wb*yb+(xb-1);

	// surrounding pixel sum
	const float surPx0 = target[nlt*3+0] + target[slt*3+0] + target[wlt*3+0] + target[elt*3+0];
	const float surPx1 = target[nlt*3+1] + target[slt*3+1] + target[wlt*3+1] + target[elt*3+1];
	const float surPx2 = target[nlt*3+2] + target[slt*3+2] + target[wlt*3+2] + target[elt*3+2];

	// constant neighbor pixel
	float conPx0 = 0.0f, conPx1 = 0.0f, conPx2 = 0.0f;
	// accumulate the background pixels
	if (mask[nlt] <= 127.0f) {
		conPx0 += background[nlb*3+0];
		conPx1 += background[nlb*3+1];
		conPx2 += background[nlb*3+2];
	}
	if (mask[slt] <= 127.0f) {
		conPx0 += background[slb*3+0];
		conPx1 += background[slb*3+1];
		conPx2 += background[slb*3+2];
	}
	if (mask[wlt] <= 127.0f) {
		conPx0 += background[wlb*3+0];
		conPx1 += background[wlb*3+1];
		conPx2 += background[wlb*3+2];
	}
	if (mask[elt] <= 127.0f) {
		conPx0 += background[elb*3+0];
		conPx1 += background[elb*3+1];
		conPx2 += background[elb*3+2];
	}

	// fill the constant value
	fixed[clt*3+0] = (4*target[clt*3+0] - surPx0 + conPx0)/4;
	fixed[clt*3+1] = (4*target[clt*3+1] - surPx1 + conPx1)/4;
	fixed[clt*3+2] = (4*target[clt*3+2] - surPx2 + conPx2)/4;
}

__global__ void PoissonImageCloningIteration(
	const float *fixed,
	const float *mask,
	const float *in,
	float *out,
	const int wt, const int ht
) {
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int clt = wt*yt+xt;

	// ignore out-of-range pixels
	if (yt >= ht or xt >= wt) {
		return;
	}

	const int yb = oy+yt, xb = ox+xt;
	const int clb = wb*yb+xb;

	// 1px spacing, using background values
	if (yt < 1 or yt >= ht-1 or xt < 1 or xt >= wt-1) {
		out[clt*3+0] = in[clb*3+0];
		out[clt*3+1] = in[clb*3+1];
		out[clt*3+2] = in[clb*3+2];
		return;
	}

	// calculate target N, S, W, E linear index
	const int nlt = wt*(yt+1)+xt;
	const int slt = wt*(yt-1)+xt;
	const int wlt = wt*yt+(xt-1);
	const int elt = wt*yt+(xt+1);

	// constant neighbor pixel
	float varPx0 = 0.0f, varPx1 = 0.0f, varPx2 = 0.0f;
	// accumulate the background pixels
	if (mask[nlt] > 127.0f) {
		varPx0 += in[nlt*3+0];
		varPx1 += in[nlt*3+1];
		varPx2 += in[nlt*3+2];
	}
	if (mask[slt] > 127.0f) {
		varPx0 += in[slt*3+0];
		varPx1 += in[slt*3+1];
		varPx2 += in[slt*3+2];
	}
	if (mask[wlt] > 127.0f) {
		varPx0 += in[wlt*3+0];
		varPx1 += in[wlt*3+1];
		varPx2 += in[wlt*3+2];
	}
	if (mask[elt] > 127.0f) {
		varPx0 += in[elt*3+0];
		varPx1 += in[elt*3+1];
		varPx2 += in[elt*3+2];
	}

	// fill the result
	out[clt*3+0] = fixed[clt*3+0] + varPx0/4;
	out[clt*3+1] = fixed[clt*3+1] + varPx1/4;
	out[clt*3+2] = fixed[clt*3+2] + varPx2/4;
}

void PoissonImageCloning(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
)
{
	// setup
	float *fixed, *buf1, *buf2;
	hipMalloc(&fixed, 3*wt*ht*sizeof(float));
	hipMalloc(&buf1, 3*wt*ht*sizeof(float));
	hipMalloc(&buf2, 3*wt*ht*sizeof(float));

	// initialize the iteration
	dim3 gdim(CeilDiv(wt, 32), CeilDiv(ht, 16)), bdim(32, 16);
	CalculateFixed<<<gdim, bdim>>>(
		background, target, mask, fixed,
		wb, hb, wt, ht, oy, ox
	);
	hipMemcpy(buf1, target, sizeof(float)*3*wt*ht, hipMemcpyDeviceToDevice);

	// iterate
	for (int i = 0; i < 1; i++) {
		PoissonImageCloningIteration<<<gdim, bdim>>>(
			fixed, mask, buf1, buf2, wt, ht
		);
		PoissonImageCloningIteration<<<gdim, bdim>>>(
			fixed, mask, buf2, buf1, wt, ht
		);
	}

	// copy the image back
	hipMemcpy(output, background, wb*hb*sizeof(float)*3, hipMemcpyDeviceToDevice);
	SimpleClone<<<dim3(CeilDiv(wt,32), CeilDiv(ht,16)), dim3(32,16)>>>(
		background, buf2, mask, output,
		wb, hb, wt, ht, oy, ox
	);

	// clean up
	hipFree(fixed);
	hipFree(buf1);
	hipFree(buf2);
}
