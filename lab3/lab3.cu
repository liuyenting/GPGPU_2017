#include "hip/hip_runtime.h"
#include "lab3.h"
#include <cstdio>
#include <cassert>

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

__global__ void SimpleClone(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
)
{
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt+xt;
	if (yt < ht and xt < wt and mask[curt] > 127.0f) {
		const int yb = oy+yt, xb = ox+xt;
		const int curb = wb*yb+xb;
		if (0 <= yb and yb < hb and 0 <= xb and xb < wb) {
			output[curb*3+0] = target[curt*3+0];
			output[curb*3+1] = target[curt*3+1];
			output[curb*3+2] = target[curt*3+2];
		}
	}
}

__global__ void CalculateFixed(
	const float *background,
	const float *target,
	const float *mask,
	float *fixed,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
) {
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int clt = wt*yt+xt;

	// ignore out-of-range pixels
	if (yt >= ht or xt >= wt) {
		return;
	}

	const int yb = oy+yt, xb = ox+xt;

	// calculate target N, S, W, E linear index
	const int nlt = wt*(yt+1)+xt;
	const int slt = wt*(yt-1)+xt;
	const int wlt = wt*yt+(xt-1);
	const int elt = wt*yt+(xt+1);
	// calculate background N, S, W, E linear index
	const int nlb = wb*(yb+1)+xb;
	const int slb = wb*(yb-1)+xb;
	const int wlb = wb*yb+(xb-1);
	const int elb = wb*yb+(xb+1);

	int npx = 4;
	// surrounding pixel sum
	float surPx0 = 0.0f, surPx1 = 0.0f, surPx2 = 0.0f;
	// constant neighbor pixel
	float conPx0 = 0.0f, conPx1 = 0.0f, conPx2 = 0.0f;
	if (yt < ht-1) {
		surPx0 += target[nlt*3+0];
		surPx1 += target[nlt*3+1];
		surPx2 += target[nlt*3+2];

		if (mask[nlt] <= 127.0f) {
			conPx0 += background[nlb*3+0];
			conPx1 += background[nlb*3+1];
			conPx2 += background[nlb*3+2];
		}
	} else {
		npx--;
	}
	if (yt > 1) {
		surPx0 += target[slt*3+0];
		surPx1 += target[slt*3+1];
		surPx2 += target[slt*3+2];

		if (mask[slt] <= 127.0f) {
			conPx0 += background[slb*3+0];
			conPx1 += background[slb*3+1];
			conPx2 += background[slb*3+2];
		}
	} else {
		npx--;
	}
	if (xt > 1) {
		surPx0 += target[wlt*3+0];
		surPx1 += target[wlt*3+1];
		surPx2 += target[wlt*3+2];

		if (mask[wlt] <= 127.0f) {
			conPx0 += background[wlb*3+0];
			conPx1 += background[wlb*3+1];
			conPx2 += background[wlb*3+2];
		}
	} else {
		npx--;
	}
	if (xt < wt-1) {
		surPx0 += target[elt*3+0];
		surPx1 += target[elt*3+1];
		surPx2 += target[elt*3+2];

		if (mask[elt] <= 127.0f) {
			conPx0 += background[elb*3+0];
			conPx1 += background[elb*3+1];
			conPx2 += background[elb*3+2];
		}
	} else {
		npx--;
	}

	// fill the constant value
	fixed[clt*3+0] = (npx*target[clt*3+0] - surPx0 + conPx0)/npx;
	fixed[clt*3+1] = (npx*target[clt*3+1] - surPx1 + conPx1)/npx;
	fixed[clt*3+2] = (npx*target[clt*3+2] - surPx2 + conPx2)/npx;
}

__global__ void PoissonImageCloningIteration(
	const float *fixed,
	const float *mask,
	const float *in,
	float *out,
	const int wt, const int ht
) {
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int clt = wt*yt+xt;

	// ignore out-of-range pixels
	if (yt >= ht or xt >= wt) {
		return;
	}

	// calculate target N, S, W, E linear index
	const int nlt = wt*(yt+1)+xt;
	const int slt = wt*(yt-1)+xt;
	const int wlt = wt*yt+(xt-1);
	const int elt = wt*yt+(xt+1);

	// total pixels
	int npx = 4;
	// constant neighbor pixel
	float varPx0 = 0.0f, varPx1 = 0.0f, varPx2 = 0.0f;
	// accumulate the background pixels
	if (yt < ht-1) {
		if (mask[nlt] > 127.0f) {
			varPx0 += in[nlt*3+0];
			varPx1 += in[nlt*3+1];
			varPx2 += in[nlt*3+2];
		}
	} else {
		npx--;
	}
	if (yt > 1) {
		if (mask[slt] > 127.0f) {
			varPx0 += in[slt*3+0];
			varPx1 += in[slt*3+1];
			varPx2 += in[slt*3+2];
		}
	} else {
		npx--;
	}
	if (xt > 1) {
		if (mask[wlt] > 127.0f) {
			varPx0 += in[wlt*3+0];
			varPx1 += in[wlt*3+1];
			varPx2 += in[wlt*3+2];
		}
	} else {
		npx--;
	}
	if (xt < wt-1) {
		if (mask[elt] > 127.0f) {
			varPx0 += in[elt*3+0];
			varPx1 += in[elt*3+1];
			varPx2 += in[elt*3+2];
		}
	} else {
		npx--;
	}
	assert(npx > 0);

	// fill the result
	out[clt*3+0] = fixed[clt*3+0] + varPx0/npx;
	out[clt*3+1] = fixed[clt*3+1] + varPx1/npx;
	out[clt*3+2] = fixed[clt*3+2] + varPx2/npx;
}

void PoissonImageCloning(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
)
{
	// setup
	float *fixed, *buf1, *buf2;
	hipMalloc(&fixed, 3*wt*ht*sizeof(float));
	hipMalloc(&buf1, 3*wt*ht*sizeof(float));
	hipMalloc(&buf2, 3*wt*ht*sizeof(float));

	// initialize the iteration
	dim3 gdim(CeilDiv(wt, 32), CeilDiv(ht, 16)), bdim(32, 16);
	CalculateFixed<<<gdim, bdim>>>(
		background, target, mask, fixed,
		wb, hb, wt, ht, oy, ox
	);
	hipMemcpy(buf1, target, sizeof(float)*3*wt*ht, hipMemcpyDeviceToDevice);

	// iterate
	for (int i = 0; i < 10000; i++) {
		PoissonImageCloningIteration<<<gdim, bdim>>>(
			fixed, mask, buf1, buf2, wt, ht
		);
		PoissonImageCloningIteration<<<gdim, bdim>>>(
			fixed, mask, buf2, buf1, wt, ht
		);
	}

	// copy the image back
	hipMemcpy(output, background, wb*hb*sizeof(float)*3, hipMemcpyDeviceToDevice);
	SimpleClone<<<dim3(CeilDiv(wt,32), CeilDiv(ht,16)), dim3(32,16)>>>(
		background, buf1, mask, output,
		wb, hb, wt, ht, oy, ox
	);

	// clean up
	hipFree(fixed);
	hipFree(buf1);
	hipFree(buf2);
}
